#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <hip/hip_runtime.h>

typedef float2 Complex;

#define THREADS 32
#define MAX_NO_OF_THREADS_PER_BLOCK 1024

const long long ARRAY_SIZE = 16; 
const long long ARRAY_BYTES = ARRAY_SIZE * sizeof(Complex);

__global__ void bit_reverse_reorder(Complex *d_rev, Complex *d_a, int s) {
	  int id = blockIdx.x * blockDim.x + threadIdx.x;
    int rev = __brev(id) >> (32-s);

    if(id < ARRAY_SIZE)
        d_rev[rev] = d_a[id];
}

__global__ void swap_real_and_imaginary(Complex *d_rev) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < ARRAY_SIZE) {
        float temp = d_rev[id].x;
        d_rev[id].x = d_rev[id].y;
        d_rev[id].y = temp;
    }
}

__device__ void inplace_fft(Complex *a, int j, int k, int m){
    
    if (j+k+m/2 < ARRAY_SIZE){
        
        Complex w, t, u;

        // w^k (w is root of unity)
        w.x = __cosf((2*M_PI*k)/m);
        w.y = -__sinf((2*M_PI*k)/m);

        // u = a[j+k]
        u.x = a[j+k].x;
        u.y = a[j+k].y;

        // t = w*a[j+k+m/2];
        t.x = w.x*a[j+k+m/2].x - w.y*a[j+k+m/2].y;
        t.y = w.x*a[j+k+m/2].y + w.y*a[j+k+m/2].x;

        // a[j+k] = u+t;
        a[j+k].x = u.x + t.x;
        a[j+k].y = u.y + t.y;

        // a[j+k+m/2] = u-t;
        a[j+k+m/2].x = u.x - t.x;
        a[j+k+m/2].y = u.y - t.y;

    }
}

__global__ void fft_outer(Complex *a, int m){
    int j = (blockIdx.x * blockDim.x + threadIdx.x)*m;
    if (j < ARRAY_SIZE){
        for (int k=0;k<m/2;k++){
            inplace_fft(a,j,k,m);
        }
    }    
}

__global__ void fft_inner(Complex *a, int j, int m){
    int k = (blockIdx.x * blockDim.x + threadIdx.x);
    if (k < m/2)
        inplace_fft(a,j,k,m);
}

int main() 
{

    //Measuring performance
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Creating files to write output to
    FILE *fptr;
    fptr = fopen("fft-opt1-output.dat", "wr");
    
    Complex h_a[ARRAY_SIZE];
    Complex h_rev[ARRAY_SIZE];

    for(int i = 0; i < ARRAY_SIZE; i++) 
    {
    	  h_a[i].x = sin((12*M_PI*i)/ARRAY_SIZE);
        h_a[i].y = 0.0;
    }
    	
    int s = (int)ceil(log2(ARRAY_SIZE));

    Complex *d_a, *d_rev, *d_rev1;

    hipMalloc((void**) &d_a, ARRAY_BYTES);
    hipMalloc((void**) &d_rev, ARRAY_BYTES);
    hipMalloc((void**) &d_rev1, ARRAY_BYTES);

    hipMemcpy(d_a, h_a, ARRAY_BYTES, hipMemcpyHostToDevice);

    //Start of performance measurement
    hipEventRecord(start);

    bit_reverse_reorder<<<(ARRAY_SIZE+THREADS-1)/THREADS, THREADS>>>(d_rev, d_a, s);
    
    hipDeviceSynchronize();

    for (int i=1;i<=s;i++){
        int m = 1 << i;
        if (m/2 < MAX_NO_OF_THREADS_PER_BLOCK){
            fft_outer<<<((ARRAY_SIZE/m)+THREADS-1)/THREADS,THREADS>>>(d_rev,m);    
        } else {
            for (int j=0;j<ARRAY_SIZE;j+=m){
                fft_inner<<<((m/2)+THREADS-1)/THREADS,THREADS>>>(d_rev,j,m);
            }
        }
    }

    hipDeviceSynchronize();

    // Beginning of inverse FFT

    swap_real_and_imaginary<<<(ARRAY_SIZE+THREADS-1)/THREADS, THREADS>>>(d_rev);

    hipDeviceSynchronize();

    bit_reverse_reorder<<<(ARRAY_SIZE+THREADS-1)/THREADS, THREADS>>>(d_rev1, d_rev, s);

    hipDeviceSynchronize();
    
    for (int i=1;i<=s;i++)
    {
        int m = 1 << i;
        if (m < sqrt(ARRAY_SIZE / 4))
        {
            fft_outer<<<((ARRAY_SIZE/m)+THREADS-1)/THREADS,THREADS>>>(d_rev1,m);    
        } 
        else 
        {
            for (int j=0;j<ARRAY_SIZE;j+=m)
            {
                fft_inner<<<((m/2)+THREADS-1)/THREADS,THREADS>>>(d_rev1,j,m);
            }
        }
    }

    swap_real_and_imaginary<<<(ARRAY_SIZE+THREADS-1)/THREADS, THREADS>>>(d_rev1);

    //End of performance measurement
    hipEventRecord(stop);

    //Block CPU execution until the event "stop" is recorded
    hipEventSynchronize(stop);

    //Print the time taken in milliseconds
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("The total time taken is %f milliseconds\n", milliseconds);

    hipMemcpy(h_rev,d_rev1,ARRAY_BYTES,hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_rev);
    hipFree(d_rev1);
    
    for (int i=0;i<ARRAY_SIZE;i++){
        printf ("%f %f\n", h_rev[i].x/ARRAY_SIZE, h_rev[i].y/ARRAY_SIZE);
    }

    // Writing output to files
    fprintf(fptr, "i\t\ta.magn\t\ta.real\t\t\ta.img\n");
    for (int i = 0; i < ARRAY_SIZE; i++)
    {
        fprintf(fptr,"%d\t\t%f\t\t%f\t\t%f\n", i, magnitude(h_rev1[i]), h_rev1[i].x, h_rev1[i].y);
    }

    return 0;
}